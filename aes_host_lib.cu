/*
 * @file aes_host_library.cu
 * @brif host library
 *
 *      Author: yebeman
 */


/*******************************************************************************
* Includes
*******************************************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <chrono>

#include "aes_config.h"
#include "aes_ui.h"
#include "aes_host_lib.h"

/*******************************************************************************
* Definitions
*******************************************************************************/

//typedefs

//constants

//macros

/*******************************************************************************
* Public Variables
*******************************************************************************/

/*******************************************************************************
* Private Variables
*******************************************************************************/

/*******************************************************************************
* Private Prototypes
*******************************************************************************/

/*******************************************************************************
* Interrupt Service Routines
*******************************************************************************/

/*******************************************************************************
* Public Functions
*******************************************************************************/

aes_host_lib_status_t aes_host_lib_init()
{

    return AES_HOST_LIB_SUCCESS;
}


// Read array in from file
// with padding by one block crunchable data chunk
aes_host_lib_status_t aes_host_lib_read_file( aes_ui_file_t *aes_ui_file, bool isPadding) 
{
    
    size_t _size = 0,
           _padded_size = 0;

    FILE* _input = fopen(aes_ui_file->name, "r");
    if (_input == NULL) {

        printf("Error opening file %s\n", aes_ui_file->name);
        return AES_HOST_LIB_FAILURE;  

    }

    // find length
    fseek(_input, 0, SEEK_END);
    _size = ftell(_input);
    rewind (_input);

    // pad per block data
    // block data = BLOCK_SIZE * BYTE_PER_THREAD
    _padded_size = _size;

    if (isPadding)
     {
        _padded_size += BLOCK_SIZE * BYTE_PER_THREAD - _size % ( BLOCK_SIZE * BYTE_PER_THREAD );
     } 


    // create a location
    aes_ui_file->content = (unsigned char *) malloc( _padded_size * sizeof(unsigned char) );
    memset(aes_ui_file->content, 0x00, _padded_size * sizeof(unsigned char) );

    // read
    if (fread(aes_ui_file->content, 1, _size, _input) != _size)
    {
        printf("Unable to read all bytes from file %s\n", aes_ui_file->name);
        return AES_HOST_LIB_FAILURE;  
    }

    aes_ui_file->padded_size = _padded_size;
    aes_ui_file->size         = _size;

    fclose (_input);

    return AES_HOST_LIB_SUCCESS;

}

aes_host_lib_status_t aes_host_lib_write_file( aes_ui_file_t *aes_ui_file) 
{

    FILE* _output = fopen(aes_ui_file->name, "w");
    if (_output == NULL) {

        printf("Error opening file %s\n", aes_ui_file->name);
        return AES_HOST_LIB_FAILURE;  

    }

    fwrite(aes_ui_file->content, sizeof(unsigned char), aes_ui_file->size, _output);
    fclose (_output);

    return AES_HOST_LIB_SUCCESS;
}





/*******************************************************************************
* Private Functions
*******************************************************************************/
