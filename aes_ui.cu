/*
 * @file aes_user_interface.h
 * @brif user interface
 *
 *      Author: yebeman
 */


/*******************************************************************************
* Includes
*******************************************************************************/
    
#include <stdio.h>

#include "aes_ui.h"

/*******************************************************************************
* Definitions
*******************************************************************************/

//typedefs

//constants

//macros

/*******************************************************************************
* Public Variables
*******************************************************************************/

/*******************************************************************************
* Private Variables
*******************************************************************************/

/*******************************************************************************
* Private Prototypes
*******************************************************************************/
static void aes_ui_help();

/*******************************************************************************
* Interrupt Service Routines
*******************************************************************************/

/*******************************************************************************
* Public Functions
*******************************************************************************/

aes_ui_status_t aes_ui_init() 
{

    // error out if its not with in the legend 
    // else perfom

    return AES_UI_SUCCESS;
}

aes_ui_status_t aes_ui(int argc, char** argv, aes_ui_t *aes_ui)
{
    
    // argc num of arguments passed
    // argv pointer array to each argument
    switch(argc-1)
    {
        case 1: 

            if ((strncmp(argv[1], "-h", strlen( "-h" )) && strncmp(argv[1], "-help", strlen( "-help" ))) == 0)
            {
                aes_ui_help();

                return AES_UI_FAILURE;

            }         
            else
            {
                // show how to use
                aes_ui_help();

                return AES_UI_FAILURE;
            }

        break;
        case 2:
            if ((strncmp(argv[1], "-test", strlen( "-test" )) && strncmp(argv[1], "-test", strlen( "-test" ))) == 0)
            {

                if ( strncmp(argv[2], "-e", strlen( "-e" ))  == 0 )
                {
                    aes_ui->encrypt = true;
                    printf("Peforming Encryption - \n");              

                    aes_ui->input_file.name = ( char *) malloc( strlen("plaintext.file") );
                    strcpy(aes_ui->input_file.name, "plaintext.file");
                    printf("Input file : %s \n", aes_ui->input_file.name);              

                    aes_ui->output_file.name = ( char *) malloc( strlen("cipher.file") );
                    strcpy(aes_ui->output_file.name, "cipher.file");
                    printf("Output file: %s \n", aes_ui->output_file.name); 

                }
                else if ( strncmp(argv[2], "-d", strlen( "-d" ))  == 0 )
                {
                    aes_ui->encrypt = false;
                    printf("Peforming Decryption - \n");              

                    aes_ui->input_file.name = ( char *) malloc( strlen("cipher.file") );
                    strcpy(aes_ui->input_file.name, "cipher.file");
                    printf("Input file : %s \n", aes_ui->input_file.name);              

                    aes_ui->output_file.name = ( char *) malloc( strlen("plaintext.file") );
                    strcpy(aes_ui->output_file.name, "plaintext.file"); 
                    printf("Output file: %s \n", aes_ui->output_file.name); 
                } 

                aes_ui->key.name = ( char *) malloc( strlen("key.file") );
                strcpy(aes_ui->key.name, "key.file"); 
                printf("Key: %s \n\n", aes_ui->key.name); 

            }   
        break;

        case 9:

            // for now loop through all 
            // save info
            for (int _index = 0; _index <= 9; ++_index)
            {

                if ( strncmp(argv[_index], "-e", strlen( "-e" ))  == 0 )
                {
                    aes_ui->encrypt = true;
                }
                else if ( strncmp(argv[_index], "-d", strlen( "-d" ))  == 0 )
                {
                    aes_ui->encrypt = false;
                }                
                else if ( strncmp(argv[_index], "-key", strlen( "-key" )) == 0)
                {
                
                    // create and save
                    aes_ui->key.name = ( char *) malloc( strlen(argv[_index + 1]) );
                    strcpy(aes_ui->key.name, argv[_index + 1]); 

//                    memcpy(aes_ui->key.name, argv[_index + 1], strlen(argv[_index + 1]) );

                }
                else if ( strncmp(argv[_index], "-out", strlen( "-out" )) == 0)
                {
                
                    // create and save
                    aes_ui->output_file.name = ( char *) malloc( strlen(argv[_index + 1]) );
                    strcpy(aes_ui->output_file.name, argv[_index + 1]); 


                   // memcpy(aes_ui->output_file.name, argv[_index + 1], strlen(argv[_index + 1]) );

                }
                else if ( strncmp(argv[_index], "-in", strlen( "-in" )) == 0)
                {
                
                    // create and save
                    aes_ui->input_file.name = ( char *) malloc( strlen(argv[_index + 1]) );
                    strcpy(aes_ui->output_file.name, argv[_index + 1]); 

                //    memcpy(aes_ui->input_file.name, argv[_index + 1], strlen(argv[_index + 1]) );

                }
                else if ( strncmp(argv[_index], "-c", strlen( "-c" )) == 0)
                {

                    if ( strncmp(argv[_index + 1], "ECB_128", strlen( "ECB_128" )) == 0 )
                    {
                    
                        aes_ui->aes_ui_cipher = ECB_128;
                    
                    }

                }

            }


        break;

        default:

            // show how to use
            aes_ui_help();

            return AES_UI_FAILURE;
        break;

    }

    return AES_UI_SUCCESS;

}

/*******************************************************************************
* Private Functions
*******************************************************************************/

static void aes_ui_help()
{

	printf("\n\r AES on DEVICE CLI Usage:\n\r");

    printf("\t%-30s%-8s",
            "-c",
            "- cipher type\n\r");

    printf("\t%-30s%-20s",
            "-d/-e",
            "- decrypt/encrypt \n\r");

    printf("\t%-30s%-20s",
            "-in/-out",
            "- input file / output file\n\r");

    printf("\t%-30s%-14s",
            "-key",
            "- secure key file \n\r");

    printf("\t%-30s%-14s",
            "-h/other",
            "- show usage  \n\n\r");

    printf("\t%-30s%-14s",
            "-test -d/-e",
            "- will do test run of decryption/encryption\n\n\r");

    printf("\n\r eg : ./aes_d -e -key key.file -c ECB_128 -in input.file -out output.file \n\r");
}

