#include "hip/hip_runtime.h"
/*
 * @file aes_engine.cu
 * @brif aes engine
 *
 *      Author: yebeman
 */


/*******************************************************************************
* Includes
*******************************************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <chrono>

#include "aes_config.h"
#include "aes_ui.h"
#include "aes_host_lib.h"
#include "aes_device_lib.h"
#include "aes_engine.h"

/*******************************************************************************
* Definitions
*******************************************************************************/

//typedefs

//constants


//macros

/*******************************************************************************
* Public Variables
*******************************************************************************/

/*******************************************************************************
* Private Variables
*******************************************************************************/
//  -- data stream
static hipStream_t cudaStream[NUM_STREAMS];

static unsigned char *input_memory_pool[NUM_STREAMS],
                     *output_memory_pool[NUM_STREAMS];

static unsigned char *d_key;

// set up timer
static hipEvent_t start[NUM_STREAMS], stop[NUM_STREAMS];
static float elapsed_time  = 0.0f;

static int seg_size = 0;

// Setup the execution configuration
static dim3 dimBlock, dimGrid;

/*******************************************************************************
* Private Prototypes
*******************************************************************************/
aes_engine_status_t aes_engine_encrypt(aes_ui_t *aes_ui);
aes_engine_status_t aes_engine_decrypt(aes_ui_t *aes_ui);

/*******************************************************************************
* Interrupt Service Routines
*******************************************************************************/

/*******************************************************************************
* Public Functions
*******************************************************************************/

aes_engine_status_t aes_engine_init()
{

    for (int _index = 0; _index < NUM_STREAMS; _index ++)
    {

        hipEventCreate(&start[_index]);
        hipEventCreate(&stop[_index]);
        hipStreamCreate(&cudaStream[_index]);
    }

    return AES_ENGINE_SUCCESS;
}



aes_engine_status_t aes_engine(aes_ui_t *aes_ui)
{

    // read input
    if ( AES_HOST_LIB_SUCCESS != aes_host_lib_read_file(&aes_ui->input_file, true) )
    {    
        // error
        return AES_ENGINE_FAILURE;
    }

    // read key
    if ( AES_HOST_LIB_SUCCESS != aes_host_lib_read_file(&aes_ui->key, false) )
    {    
        // error
        return AES_ENGINE_FAILURE;
    }

    // create memory pool for output
    // on AES - ECB padding is up to the user
    aes_ui->output_file.content = (unsigned char *) malloc(aes_ui->input_file.padded_size * sizeof(unsigned char) );
    memset(aes_ui->output_file.content, 0x00,  aes_ui->input_file.padded_size * sizeof(unsigned char));
    aes_ui->output_file.size    = aes_ui->input_file.padded_size; // output will be have padd ending 

    // 6 block/sm * 256 thread/ block * 20 = 30720 threads
    // 30720 threads * 16 byte = 491,520 byte = 491Kbyte
    // aes_ui->input_file.padded_size is a multiple of 4096 or 256*16
    seg_size = aes_ui->input_file.padded_size / NUM_STREAMS ;

    // create memory
    for (int _index = 0; _index < NUM_STREAMS; _index ++)
    {
        hipMalloc((void**) &input_memory_pool[_index],  seg_size * sizeof(unsigned char));
        hipMemset(input_memory_pool[_index],   0x00, seg_size * sizeof(unsigned char));

        hipMalloc((void**) &output_memory_pool[_index], seg_size * sizeof(unsigned char));
        hipMemset(output_memory_pool[_index],  0x00, seg_size * sizeof(unsigned char));
    }

    hipMalloc((void**) &d_key, 16 * sizeof(unsigned char));
    hipMemcpy(d_key, 
               aes_ui->key.content,  
               16 * sizeof(unsigned char), 
               hipMemcpyHostToDevice);

    // set block size
    dimBlock.x = BLOCK_SIZE;   
    dimBlock.y = 1;
    dimBlock.z = 1;

    // set grid size
    dimGrid.x = ceil( float (seg_size)  / float (BLOCK_SIZE * 16) );
    dimGrid.y = 1;
    dimGrid.z = 1;

    if (aes_ui->encrypt)
    {
        aes_engine_encrypt(aes_ui);
    }
    else 
    {
        aes_engine_decrypt(aes_ui);
    }

    hipDeviceSynchronize();

    // add time take in all the streams
    float _elapsed = 0.0f;
    for (int _index = 0; _index < NUM_STREAMS; _index ++)
    {

        hipEventSynchronize(stop[_index]);

        hipEventElapsedTime(&_elapsed, start[_index], stop[_index]);

        elapsed_time+=_elapsed;
    }

    printf("\nElapsed time: %f ms\n\n", elapsed_time);

       // write file
    aes_host_lib_write_file(&aes_ui->output_file);

    return AES_ENGINE_SUCCESS;
}



/*******************************************************************************
* Private Functions
*******************************************************************************/

aes_engine_status_t aes_engine_encrypt(aes_ui_t *aes_ui)
{

    for (int _index = 0; _index < NUM_STREAMS; _index++)
    {

        hipEventRecord(start[_index], cudaStream[_index]);

        hipMemcpyAsync(input_memory_pool[_index], 
                        aes_ui->input_file.content + _index * seg_size,
                        seg_size * sizeof(unsigned char),
                        hipMemcpyHostToDevice,
                        cudaStream[_index]);

        aes_device_lib_encrypt<<<dimGrid,dimBlock,0,cudaStream[_index]>>>(input_memory_pool[_index],
                                                                          output_memory_pool[_index],
                                                                          d_key,
                                                                          seg_size);

        hipMemcpyAsync(aes_ui->output_file.content + _index * seg_size,
                                output_memory_pool[_index],
                                seg_size * sizeof(unsigned char),
                                hipMemcpyDeviceToHost,
                                cudaStream[_index]);

        hipEventRecord(stop[_index], cudaStream[_index]);

    }

    return AES_ENGINE_SUCCESS;

}


aes_engine_status_t aes_engine_decrypt(aes_ui_t *aes_ui)
{
    for (int _index = 0; _index < NUM_STREAMS; _index++)
    {

        hipEventRecord(start[_index], cudaStream[_index]);

        hipMemcpyAsync(input_memory_pool[_index],
                        aes_ui->input_file.content + _index * seg_size,
                        seg_size * sizeof(unsigned char),
                        hipMemcpyHostToDevice,
                        cudaStream[_index]);

        aes_device_lib_decrypt<<<dimGrid,dimBlock,0,cudaStream[_index]>>>(input_memory_pool[_index],
                                                                          output_memory_pool[_index],
                                                                          d_key,
                                                                          seg_size);

        hipMemcpyAsync(aes_ui->output_file.content + _index * seg_size,
                        output_memory_pool[_index],
                        seg_size * sizeof(unsigned char),
                        hipMemcpyDeviceToHost,
                        cudaStream[_index]);

        hipEventRecord(stop[_index], cudaStream[_index]);
    }

    return AES_ENGINE_SUCCESS;
}